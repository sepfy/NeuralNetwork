#include "hip/hip_runtime.h"
#include "layers.h"

__global__ void relu_activate_gpu_kernel(float *input, float *output, int size) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if( index > size)
    return;
  output[index] = input[index]*(input[index] >=0);
}

void Activation::relu_activate_gpu() {

  int grid = batch*((N-1)/256 + 1);
    
  relu_activate_gpu_kernel<<<grid, 256>>>(input, output, batch*N);
  check_error(hipGetLastError());
}


__global__ void relu_backward_gpu_kernel(float *m_delta, float *delta, float *input, float *cut, int size) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if( index > size)
    return;
  m_delta[index] = (cut[index] + delta[index])*(input[index] >= 0);
}



void Activation::relu_backward_gpu(float *delta) {

  int grid = batch*((N-1)/256 + 1);
  relu_backward_gpu_kernel<<<grid, 256>>>(m_delta, delta, input, cut, batch*N);
  check_error(hipGetLastError());
}


__global__ void leaky_activate_gpu_kernel(float *input, float *output, int size) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if( index > size)
    return;
  output[index] = (input[index] >= 0 ? input[index] : 0.1*input[index]);
}

void Activation::leaky_activate_gpu() {

  leaky_activate_gpu_kernel<<<default_grid(batch*N), BLOCK>>>(input, output, batch*N);
  check_error(hipGetLastError());
}


__global__ void leaky_backward_gpu_kernel(float *m_delta, float *delta, float *input, float *cut, int size) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if( index > size)
    return;
  m_delta[index] = (cut[index] + delta[index])*(input[index] >= 0 ? 1.0 : 0.1);
}



void Activation::leaky_backward_gpu(float *delta) {

  leaky_backward_gpu_kernel<<<default_grid(batch*N), BLOCK>>>(m_delta, delta, input, cut, batch*N);
  check_error(hipGetLastError());
}

__global__ void softmax_kernel(float *input, float *output, int N) {

    int i = threadIdx.x;
    float tmp = 0;
    float max = 0;
    for(int j = 0; j < N; j++)
      if(input[i*N+j] > max)
        max = input[i*N+j];

    for(int j = 0; j < N; j++) {
      output[i*N+j] = exp(input[i*N+j] - max);
      tmp += output[i*N+j];
    }
    for(int j = 0; j < N; j++)
      output[i*N+j] /= tmp;

}

void SoftmaxWithCrossEntropy::forward_gpu() {

  softmax_kernel<<<1, batch>>>(input, output, N);
  check_error(hipGetLastError());
}

