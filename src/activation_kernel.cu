#include "hip/hip_runtime.h"
#include "layers.h"

__global__ void relu_activate_gpu_kernel(float *input, float *output, int size) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if( index > size)
    return;
  output[index] = input[index]*(input[index] >=0);
}

void Activation::relu_activate_gpu() {

  int grid = batch*((N-1)/256 + 1);
    
  relu_activate_gpu_kernel<<<grid, 256>>>(input, output, batch*N);
  check_error(hipGetLastError());
}


__global__ void relu_backward_gpu_kernel(float *m_delta, float *delta, float *input, float *cut, int size) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if( index > size)
    return;
  m_delta[index] = (cut[index] + delta[index])*(input[index] >= 0);
}



void Activation::relu_backward_gpu(float *delta) {

  int grid = batch*((N-1)/256 + 1);
  relu_backward_gpu_kernel<<<grid, 256>>>(m_delta, delta, input, cut, batch*N);
  check_error(hipGetLastError());
}


__global__ void leaky_activate_gpu_kernel(float *input, float *output, int size) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if( index > size)
    return;
  output[index] = (input[index] >= 0 ? input[index] : 0.1*input[index]);
}

void Activation::leaky_activate_gpu() {

  leaky_activate_gpu_kernel<<<default_grid(batch*N), BLOCK>>>(input, output, batch*N);
  check_error(hipGetLastError());
}


__global__ void leaky_backward_gpu_kernel(float *m_delta, float *delta, float *input, float *cut, int size) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if( index > size)
    return;
  m_delta[index] = (cut[index] + delta[index])*(input[index] >= 0 ? 1.0 : 0.1);
}



void Activation::leaky_backward_gpu(float *delta) {

  leaky_backward_gpu_kernel<<<default_grid(batch*N), BLOCK>>>(m_delta, delta, input, cut, batch*N);
  check_error(hipGetLastError());
}


void Activation::forward_gpu() {

   switch(activation) {
    case RELU:
      relu_activate_gpu();
    case LEAKY:
      leaky_activate_gpu();
  }
}

void Activation::backward_gpu(float *delta) {

  switch(activation) {
    case RELU:
      relu_backward_gpu(delta);
    case LEAKY:
      leaky_backward_gpu(delta);
  }
}

__global__ void softmax_kernel(float *input, float *output, int N) {

    int i = threadIdx.x;
    float tmp = 0;
    float max = 0;
    for(int j = 0; j < N; j++)
      if(input[i*N+j] > max)
        max = input[i*N+j];

    for(int j = 0; j < N; j++) {
      output[i*N+j] = exp(input[i*N+j] - max);
      tmp += output[i*N+j];
    }
    for(int j = 0; j < N; j++)
      output[i*N+j] /= tmp;

}

void SoftmaxWithCrossEntropy::forward_gpu() {

  softmax_kernel<<<1, batch>>>(input, output, N);
  check_error(hipGetLastError());
}

void SoftmaxWithCrossEntropy::backward_gpu(float *delta) {
  float alpha = 1.0/(float)batch;
  size_t size = sizeof(float)*batch*N;
  hipError_t status = hipMemset(m_delta, 0, size);
  check_error(status);

  hipblasSaxpy(gpu_handle(), batch*N, &alpha, output, 1, m_delta, 1);
  check_error(hipGetLastError());

  alpha = -1.0/(float)batch;
  hipblasSaxpy(gpu_handle(), batch*N, &alpha, delta, 1, m_delta, 1);
  check_error(hipGetLastError());
}


