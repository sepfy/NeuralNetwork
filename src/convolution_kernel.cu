#include "hip/hip_runtime.h"
#include "layers.h"

__global__ void bias_add_kernel(float *output, float *bias,
                         int batch, int im_size, int channel) {

    int i = threadIdx.x;
    int b = blockIdx.x;
    for(int j = 0; j < channel; j++)
      output[b*im_size*channel+i*channel+j] += bias[j];

}

void Convolution::bias_add_gpu() {

  int size = out_w*out_h;
  bias_add_kernel<<<batch, size>>>(output, bias, batch, size, FC);
  check_error(hipGetLastError());
}




