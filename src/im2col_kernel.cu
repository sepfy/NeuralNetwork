#include "hip/hip_runtime.h"
#include "layers.h"

__global__ void im2col_gpu_kernel(int W, int H, int C, int FW, int FH, int FC,
            int stride, int pad, float *im, float *col) {

  int k = blockIdx.x;
  int i = threadIdx.x;
  int j = threadIdx.y;
  int out_w = (W + 2*pad - FW)/stride + 1;
  int out_col = FH*FW*C;
  int offset_w, offset_h, c_im;
  int im_row, im_col;

  c_im = k % C;
  offset_w = k / C % FW;
  offset_h = k / C / FW;
  
  im_row = offset_h + i*stride;
  im_col = offset_w + j*stride;

  int col_idx = (i*out_w + j)*out_col + k;
  im_row -= pad;
  im_col -= pad;

  if(im_row < 0 || im_col < 0 ||
     im_row >= H || im_col >= W)
     col[col_idx] = 0.0;
   else {
      int im_idx = C*(im_row*W + im_col) + c_im;
      col[col_idx] = im[im_idx];
   }
}

void im2col_gpu(int W, int H, int C, int FW, int FH, int FC,
            int stride, int pad, float *im, float *col) {

  int out_col = FH*FW*C;
  int out_w = (W + 2*pad - FW)/stride + 1;
  int out_h = (H + 2*pad - FH)/stride + 1;

  dim3 d = {(unsigned int)out_h, (unsigned int)out_w, 1};
  im2col_gpu_kernel<<<out_col, d>>>(W, H, C, FW, FH, FC, stride, pad, im ,col);
  check_error(hipGetLastError());
}


__global__ void col2im_gpu_kernel(int W, int H, int C, int FW, int FH, int FC,
            int stride, int pad, float *im, float *col) {

  int k = blockIdx.x;
  int i = threadIdx.x;
  int j = threadIdx.y;
  int out_w = (W + 2*pad - FW)/stride + 1;
  int out_col = FH*FW*C;
  int offset_w, offset_h, c_im;
  int im_row, im_col;

  c_im = k % C;
  offset_w = k / C % FW;
  offset_h = k / C / FW;

  im_row = offset_h + i*stride;
  im_col = offset_w + j*stride;

  int col_idx = (i*out_w + j)*out_col + k;
  im_row -= pad;
  im_col -= pad;

  if(im_row < 0 || im_col < 0 ||
     im_row >= H || im_col >= W) {}
  else {
    int im_idx = C*(im_row*W + im_col) + c_im;
    im[im_idx] = col[col_idx];
  }

}

void col2im_gpu(int W, int H, int C, int FW, int FH, int FC,
            int stride, int pad, float *im, float *col) {

  int out_col = FH*FW*C;
  int out_w = (W + 2*pad - FW)/stride + 1;
  int out_h = (H + 2*pad - FH)/stride + 1;
  dim3 d = {(unsigned int)out_h, (unsigned int)out_w, 1};
  col2im_gpu_kernel<<<out_col, d>>>(W, H, C, FW, FH, FC, stride, pad, im ,col);
  check_error(hipGetLastError());
}

#if 0
__global__ void col2im_gpu_kernel(int W, int H, int C, int FW, int FH, int FC,
            int stride, int pad, float *im, float *col) {

  int k = blockIdx.x;
  int i = threadIdx.x;
  int j = threadIdx.y;
  int im_idx = (i*W + j)*C + k;

  i += pad;
  j += pad;
  int out_w_start = (j < FW) ? 0 : (j - FW)/stride + 1;
  int out_h_start = (i < FH) ? 0 : (i - FH)/stride + 1;

  int out_h = (H + 2*pad - FH)/stride + 1;
  int out_w = (W + 2*pad - FW)/stride + 1;
  int out_col = FW*FH*C;
 
  int out_h_end = i/stride + 1;
  if(out_h_end > out_h)
    out_h_end = out_h;

  int out_w_end = j/stride + 1;
  if(out_w_end > out_w)
    out_w_end = out_w;

  int h, w;
  im[im_idx] = 0.0;
  for(h = out_h_start; h < out_h_end; h++) {
    for(w = out_w_start; w < out_w_end; w++) {
      int offset_w = (j < FW) ? j : (FW - 1);
      int offset_h = (i < FH) ? i : (FH - 1);
      int col_h = (offset_h - h + out_h_start)%FH;
      int col_w = (offset_w - w + out_w_start)%FW;
      int col_idx = (h*out_w + w)*out_col + ((col_h*FW + col_w)*C) + k;
      im[im_idx] = col[col_idx];
    }
  }

}

void col2im_gpu(int W, int H, int C, int FW, int FH, int FC,
            int stride, int pad, float *im, float *col) {

  dim3 d = {(unsigned int)H, (unsigned int)W, 1};
  col2im_gpu_kernel<<<C, d>>>(W, H, C, FW, FH, FC, stride, pad, im ,col);
  check_error(hipGetLastError());
}
#endif
