#include "hip/hip_runtime.h"
#include "layers.h"

/*
__global__ void row_sum_gpu_kernel(float *A, float *B) {

  int j = blockIdx.x;
  int i = threadIdx.x;

  B
  for(int j = 0; j < M; j++)
    for(int i = 0; i < N; i++)
      B[j] += A[i*M+j];
}



void row_sum_gpu(int N, int M, float *A, float *B) {

  for(int i = 0; i < M; i++)
    hipblasSasum(gpu_handle(), N, (A + i*M), 1, (B + i));
  //row_sum_gpu_kernel<<<N, M>>>(n, x, grad_x, m_x, v_x, a.beta1, a.beta2, m_lr, a.epsilon);
  hipDeviceSynchronize();

}
*/

__global__ void row_sum_gpu_kernel(float *A, float *B, int N, int M) {
    
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index > M)
    return;
  int i = 0;
  B[index] = 0.0;
  for(i = 0; i < N; i++)
    B[index] += A[i*M + index];

}
/*
void col_sum(int N, int M, float *A, float *B) {
  memset(B, 0, M*sizeof(float));
  for(int i = 0; i < N; i++)
    for(int j = 0; j < M; j++)
      B[i] += A[i*M+j];
}


void row_sum(int N, int M, float *A, float *B) {
  memset(B, 0, N*sizeof(float));
  for(int j = 0; j < M; j++)
    for(int i = 0; i < N; i++)
      B[j] += A[i*M+j];
}

*/

void row_sum_gpu(int N, int M, float *A, float *B) {

    row_sum_gpu_kernel<<<default_grid(N), BLOCK>>>(A, B, N, M);
    check_error(hipGetLastError());
}


__global__ void col_sum_gpu_kernel(float *A, float *B, int N, int M) {
   
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index > N)
    return;
  int i = 0;
  B[index] = 0.0;
  for(i = 0; i < N; i++)
    B[index] += A[index*M + i];
/* 
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index > M)
    return;
  int i = 0;
  for(i = 0; i < M; i++)
    B[index] += A[index*M + i];
*/
}


void col_sum_gpu(int N, int M, float *A, float *B) {

/*
  int grid = (N-1)/BLOCK + 1;
  col_sum_gpu_kernel<<<grid, BLOCK>>>(A, B, N, M);
  check_error(hipGetLastError());
*/
  hipMemset(B, 0, sizeof(float)*M);
  check_error(hipGetLastError());
  float alpha = 1.0;
  float beta = 0.0;
  float *e = malloc_gpu(N);
  //hipMemset(e, 1.0, sizeof(float)*N);
  memset_gpu(e, 1, N);
  check_error(hipGetLastError());
  hipblasSgemv(gpu_handle(), HIPBLAS_OP_N, M, N, &alpha, A, M, e, 1, &beta, B, 1);
  check_error(hipGetLastError());
  hipFree(e);
}

/*
void col_sum_gpu(int N, int M, float *A, float *B) {

  memset(B, 0, N*sizeof(float));
  float alpha = 1.0;
  float beta = 0.0;
  float *e = malloc_gpu(N);
  for(int i = 0; i < N; i++)
    e[i] = 1.0;
  hipblasSgemv(gpu_handle(), HIPBLAS_OP_T, M, N, &alpha, A, M, e, 1, &beta, B, 1);
  hipDeviceSynchronize();
}
*/

__global__ void bias_add_kernel1(float *output, float *bias,
                         int batch, int size, int channel) {

    int i = threadIdx.x;
    int b = blockIdx.x;
    for(int j = 0; j < channel; j++)
      output[b*size*channel+i*channel+j] += bias[j];

}

// TODO: Integrate the bias add of convolution and connected.
void bias_add_gpu(float *output, float *bias, int batch, int size, int c) {

  bias_add_kernel1<<<batch, size>>>(output, bias, batch, size, c);
  check_error(hipGetLastError());
}

__global__ void elementwise_mul_gpu_kernel(float *A, float *B, float *C, int N) {

  int index = (blockIdx.x)*blockDim.x + threadIdx.x;
  if(index > N) return;
  C[index] = A[index] + B[index];
}

void elementwise_mul_gpu(float *A, float *B, float *C, int N) {

  elementwise_mul_gpu_kernel<<<default_grid(N),BLOCK>>>(A, B, C, N);
  check_error(hipGetLastError());

}

