#include "hip/hip_runtime.h"
#include "layer/batchnorm.h"
#include "blas.h"

void Batchnorm::Init() {

  mean = malloc_gpu(channel_);
  var  = malloc_gpu(channel_);
  running_mean = malloc_gpu(channel_);
  running_var  = malloc_gpu(channel_);

  std = malloc_gpu(channel_);

  normal = malloc_gpu(batch*spatial_*channel_);
  output = malloc_gpu(batch*spatial_*channel_);


  delta_ = malloc_gpu(batch*spatial_*channel_);
  xc = malloc_gpu(batch*channel_);

  dxn = malloc_gpu(batch*spatial_*channel_);
  dxc = malloc_gpu(batch*spatial_*channel_);
  dvar = malloc_gpu(channel_);
  dstd = malloc_gpu(channel_);
  dmu = malloc_gpu(channel_);

  gamma = malloc_gpu(channel_);
  beta = malloc_gpu(channel_);
  dgamma = malloc_gpu(channel_);
  dbeta = malloc_gpu(channel_);
  m_gamma = malloc_gpu(channel_);
  m_beta = malloc_gpu(channel_);
  v_gamma = malloc_gpu(channel_);
  v_beta = malloc_gpu(channel_);

  memset_gpu(gamma, 1.0, channel_);
}

/*
__global__ void mean_gpu_kernel(float *input, float *mean, float batch, int n_) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index >= n_)
    return;
  
  mean[index] = 0.0;
  int i;
  for(i = 0; i < batch; i++)
    mean[index] += input[i*n_ + index];

  mean[index] /= batch;

}
*/
__global__ void mean_gpu_kernel(float *input, float *mean, int batch, int spatial, int channel) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index >= channel)
    return;

  mean[index] = 0.0;

  for(int i = 0; i < batch; ++i)
    for(int j = 0; j < spatial; ++j)
      mean[index] += input[(i*spatial + j)*channel + index];

  mean[index] /= (float)(batch*spatial);

}


void Batchnorm::GetMean() {

  mean_gpu_kernel<<<default_grid(channel_), BLOCK>>>(input, mean, batch, spatial_, channel_);
  //mean_gpu_kernel<<<default_grid(n_), BLOCK>>>(input, mean, batch, n_);  
  check_error(hipGetLastError());

}

/*
__global__ void calc_xc(float *input, float *mean, float *xc) {

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.x; 
}
*/
/*
__global__ void variance_gpu_kernel(float *input, float *mean, float *var, float batch, int n_) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index >= n_)
    return;
  
  var[index] = 0.0;
  int i;

  //xc[i] = (pow(input[i] - mean[j], 2.0));
  for(i = 0; i < batch; i++)
    var[index] += pow(input[i*n_ + index] - mean[index], 2.0);

  var[index] /= batch;


}
*/

__global__ void variance_gpu_kernel(float *input, float *mean, float *var, int batch, int spatial, int channel) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index >= channel)
    return;
  
  var[index] = 0.0;

  for(int i = 0; i < batch; ++i)
    for(int j = 0; j < spatial; ++j)
      var[index] += pow(input[(i*spatial + j)*channel + index] - mean[index], 2.0);

  var[index] /= (float)(batch*spatial);


}




void Batchnorm::GetVariance() {

  variance_gpu_kernel<<<default_grid(channel_), BLOCK>>>(input, mean, var, batch, spatial_, channel_);  
  //variance_gpu_kernel<<<default_grid(n_),BLOCK>>>(input, mean, var, batch, n_);  
  check_error(hipGetLastError());

}
/*
__global__ void Normalize_gpu_kernel(float *normal, float *input, float *mean,  float *var, float epsilon) {

  int j = blockIdx.x;
  int index = gridDim.x*threadIdx.x + blockIdx.x;
  normal[index] = (input[index] - mean[j])/pow(var[j] + epsilon, 0.5);

}
*/
__global__ void get_running_variable(float momentum, float *running_x, float *x, int n) {

  int j = blockIdx.x*blockDim.x + threadIdx.x;
  if(j >= n) return;
  running_x[j] = momentum*running_x[j] + (1.0 - momentum)*x[j];

}

__global__ void normalize_gpu_kernel(float *normal, float *input, float *mean,  float *var, float epsilon, int n, int channel) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index >= n)
    return;

  int c = index%channel;
  normal[index] = (input[index] - mean[c])/pow(var[c] + epsilon, 0.5);
}

void Batchnorm::Normalize() {

  if(train_flag_) {

    int n = batch*spatial_*channel_;
    normalize_gpu_kernel<<<default_grid(n), BLOCK>>>(normal, input, mean, var, epsilon, n, channel_);
    check_error(hipGetLastError());

    get_running_variable<<<default_grid(channel_), BLOCK>>>(
		    momentum, running_mean, mean, channel_);
    check_error(hipGetLastError());

    get_running_variable<<<default_grid(channel_), BLOCK>>>(
		    momentum, running_var, var, channel_);
    check_error(hipGetLastError());


	  /*
    Normalize_gpu_kernel<<<n_, batch>>>(normal, input, mean, var, epsilon);
    check_error(hipGetLastError());

    get_running_variable<<<default_grid(n_), BLOCK>>>(
		    momentum, running_mean, mean, n_);
    check_error(hipGetLastError());

    get_running_variable<<<default_grid(n_), BLOCK>>>(
		    momentum, running_var, var, n_);
    check_error(hipGetLastError());
    */
  }
  else {
    //Normalize_gpu_kernel<<<n_, batch>>>(
	//	    normal, input, running_mean, running_var, epsilon);
    //check_error(hipGetLastError());
    int n = batch*spatial_*channel_;
    normalize_gpu_kernel<<<default_grid(n), BLOCK>>>(normal, input, mean, var, epsilon, n, channel_);
    check_error(hipGetLastError());
  }
}

__global__ void ScaleAndShift_gpu_kernel(float *output, float *normal, float *gamma, float *beta) {

  int j = blockIdx.x;
  int index = gridDim.x*threadIdx.x + blockIdx.x;
  output[index] = gamma[j]*normal[index] + beta[j];

}

__global__ void scale_and_shift_gpu_kernel(float *output, float *normal, float *gamma, float *beta, int n, int channel) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index >= n)
    return;

  int c = index%channel;
  output[index] = gamma[c]*normal[index] + beta[c];
}


void Batchnorm::ScaleAndShift() {

  int n = batch*spatial_*channel_;
  scale_and_shift_gpu_kernel<<<default_grid(n), BLOCK>>>(output, normal, gamma, beta, n, channel_);
  check_error(hipGetLastError());

}

void Batchnorm::Forward() {

  GetMean();
  GetVariance();
  Normalize();
  ScaleAndShift();
}



__global__ void cal_mdelta_kernel(float *delta_, float *dxc, float *dmu, 
		int batch, int spatial, int channel) {


  int index = blockDim.x*blockIdx.x + threadIdx.x;
  if(index >= (batch*spatial*channel)) return;
  int i = index%channel;
  delta_[index] = dxc[index] - dmu[i]/(float)(batch*spatial);

}


__global__ void calc_dxc2(float *dxc, float *input, float *mean, float *dvar,
		int batch, int spatial, int channel) {

  int index = blockDim.x*blockIdx.x + threadIdx.x;
  if(index >= (batch*spatial*channel)) return;
  int i = index%channel;
  float tmp = (float)(batch*spatial);

  dxc[index] += (2.0/tmp)*(input[index] - mean[i])*dvar[i];
}



__global__ void get_variance_delta_kernel(float *dvar, float *var, int channel) {
  
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(index >= channel) return;
  float epsilon = 1.0e-7;
  dvar[index] = 0.5*dvar[index]/pow(var[index] + epsilon, 0.5);
}


__global__ void cal_dx(float *dxn, float *dxc, float *gamma, float *delta, float *var, float *input, float *mean, int n, int channel) {

  int index = blockDim.x*blockIdx.x + threadIdx.x;
  if(index >= n) return;
  int i = index%channel;
  float epsilon = 1.0e-7;

  float tmp = gamma[i]*delta[index];
  dxn[index] = -1.0*tmp*(input[index] - mean[i])/(var[i] + epsilon);
  dxc[index] = tmp/pow(var[i] + epsilon, 0.5);
}




void Batchnorm::Backward(float *delta) {
  col_sum_gpu(batch*spatial_, channel_, delta, dbeta);

  elementwise_mul_gpu(normal, delta, normal, batch*spatial_*channel_);
  col_sum_gpu(batch*spatial_, channel_, normal, dgamma);
  int n = batch*spatial_*channel_;

  cal_dx<<<default_grid(n), BLOCK>>>(dxn, dxc, gamma, delta, var, input, mean, n, channel_);
  check_error(hipGetLastError());
  col_sum_gpu(batch*spatial_, channel_, dxn, dvar);
  get_variance_delta_kernel<<<default_grid(channel_), BLOCK>>>(dvar, var, channel_);

 
  calc_dxc2<<<default_grid(n), BLOCK>>>(dxc, input, mean, dvar, batch, spatial_, channel_);
  check_error(hipGetLastError());
  
  col_sum_gpu(batch*spatial_, channel_, dxc, dmu);
  
  cal_mdelta_kernel<<<default_grid(n), BLOCK>>>(delta_, dxc, dmu, batch, spatial_, channel_);
  check_error(hipGetLastError());
}



void Batchnorm::Update(UpdateArgs a) {

  if(a.adam) {
    adam_gpu(channel_, gamma, dgamma, m_gamma, v_gamma, a);
    adam_gpu(channel_, beta, dbeta, m_beta, v_beta, a);
  }
  else {
    momentum_gpu(channel_, gamma, dgamma, v_gamma, a);
    momentum_gpu(channel_, beta, dbeta, v_beta, a);
  }
}

